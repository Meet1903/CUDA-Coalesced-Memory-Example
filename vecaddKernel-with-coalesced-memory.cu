
#include <hip/hip_runtime.h>
// This Kernel adds two Vectors A and B in C on GPU
// using coalesced memory access.

__global__ void AddVectors(const float *A, const float *B, float *C, int N)
{
    int threadIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = blockDim.x * gridDim.x;
    int i;
    for (i = threadIndex; i < totalThreads * N; i = i + totalThreads) {
        C[i] = A[i] + B[i];
    }
}
